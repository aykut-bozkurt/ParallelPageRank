#include "hip/hip_runtime.h"
#include <thrust/reduce.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#include <math.h>

#include <stdio.h>

#include <stdlib.h>

#include <iostream>
#include <fstream>

#include <string>

#include <exception>

#include <time.h>

#include <queue>
#include <map>
#include <vector>

 
#define ALPHA 0.2
#define EPSILON 0.000001

using namespace std;

typedef struct CSRMatrix matrix;

struct CSRMatrix {
   int n_nodes;
   int n_edges;
   int * rowstarts;
   int * colindices;
   float * values;
   vector<string> *node_list; 
};


struct diffabs_functor {

	diffabs_functor(){}

	__host__ __device__
		float operator()(const float& x, const float& y) const {
			return abs(x-y);
		}
};

// x = how many nonzero values arethere in current row
// y = how many nonzero values arethere in the next row
struct matrix_vec_mul_functor {
	float alpha;
	thrust::device_ptr<float> rx;
	thrust::device_ptr<int> colindicesx;
	thrust::device_ptr<float> valuesx;
	
	matrix_vec_mul_functor(float _alpha, thrust::device_vector<float> &r_x, thrust::device_vector<int> &colindices_x, thrust::device_vector<float> &values_x) : alpha(_alpha) {
		rx = &r_x[0];
		colindicesx = &colindices_x[0];
		valuesx = &values_x[0];
	}
	__host__ __device__
		float operator()(const int& x, const int& y) const {
			int i;
			float sum = 0;
			for(i = x; i < y; i++){
				sum += valuesx[i] * rx[colindicesx[i]];
			}
			return alpha * sum;
		}
};


matrix * constructCSRMatrix();

void normalize(matrix *P);

void printtop5rank(float *r, matrix *P, int N);

 int main(int argc, char* argv[]) {
    // transition probabilities matrix
    matrix *P;
    
    // iteration variables
    int i, step;

    // difference that is compared to epsilon at each iteration
    float totalDiff = 0;
    int N;

    // read graph.txt and construct CSR matrix 
    P = constructCSRMatrix();

    // total number of vertices
    N = P->n_nodes;
   
    // current and next rank matrices
    float * r = (float*)calloc(N, sizeof(float));
    float * nextR = (float*)calloc(N, sizeof(float));

    // normalize P matrix values (each column sum = 1.0 total probability)
    normalize(P);

    /* Some initializations */
    for (i=0; i<N; i++) {
        r[i] = 1.0/N;
    }
	
    // transfer P related matrices to device
    thrust::device_vector<int> colindices_x(P->colindices, P->colindices+P->rowstarts[N]);
    thrust::device_vector<float> values_x(P->values, P->values+P->rowstarts[N]);
    thrust::device_vector<int> rowstarts_x(P->rowstarts, P->rowstarts+N+1);
	
    // transfer r and nextR to device
    thrust::device_vector<float> r_x(r, r+N);
    thrust::device_vector<float> nextR_x(nextR, nextR+N);

    // create other operational matrices in device
    thrust::device_vector<float> diffabs(N);
    thrust::device_vector<float> one_minus_alpha_over_n(N);
    thrust::device_vector<float> row_col_mul(N);
	
    time_t start,end;
    time (&start);
    step = 1;
    printf("Total nonzero value is %d ",P->rowstarts[N]);
    while(1){
	thrust::fill(diffabs.begin(), diffabs.end(), 0);

        // calculate nextR = alpha*P*r_x + (1-alpha)*r_x (P in CSR format)
	thrust::transform(rowstarts_x.begin(), rowstarts_x.end()-1, rowstarts_x.begin()+1, nextR_x.begin(), matrix_vec_mul_functor(ALPHA, r_x, colindices_x, values_x));
       
	thrust::fill(one_minus_alpha_over_n.begin(), one_minus_alpha_over_n.end(), (1-ALPHA)/N);
	thrust::transform(nextR_x.begin(), nextR_x.end(), one_minus_alpha_over_n.begin(), nextR_x.begin(), thrust::plus<float>());
	
	// calculate totaldiff in this step
	thrust::transform(r_x.begin(), r_x.end(), nextR_x.begin(), diffabs.begin(), diffabs_functor());
	totalDiff = thrust::reduce(diffabs.begin(), diffabs.end(), (float)0, thrust::plus<float>());
		
        printf("Step: %d\n", step);
        printf("Difference: %.6f\n", totalDiff);
        if(totalDiff <= EPSILON) break;
		
	// update r_x and nextR_x
	thrust::copy_n(nextR_x.begin(), N, r_x.begin());
	thrust::fill(nextR_x.begin(), nextR_x.end(), 0);
        
	// update iteration variables
	totalDiff = 0;	
	step++;
    } // end of while (algorithm converged)
	
    time (&end);
    printf("Operation took %.9f secs on GPU.\n", difftime (end,start));

    // transfer to host
    thrust::copy(r_x.begin(), r_x.end(), r);

    // output top 5 ranks
    printtop5rank(r, P, N);

    // free unused heap memory
    free(P->rowstarts);
    free(P->colindices);
    free(P->values);
    free(P);
    free(r);
    free(nextR);	
 }

matrix * constructCSRMatrix(){
	map<string,int> index_map;
	map<int, vector<int>* > adj_map;
	
	int first_index, second_index;
	int last_index = 0;
	vector<int>* neighbours;
	vector<string>* node_list = new vector<string>();

	ifstream myfile ("graph.txt");

	int steps = 0;
	string first_node(26,'0');
	string second_node(26,'0');
	string token(26,'0');
	while(!myfile.eof())
	{
    		for(int i=0;i<2;i++)
    		{
			myfile >> token;
        		if(i == 0) {
				first_node = token;
			}
			else {
				second_node = token;
			}
    		}

		try {
			first_index = index_map.at(first_node); 
		}
		catch (exception e){
			
			index_map.insert(pair<string, int>(first_node, last_index));
			
			first_index = last_index;
			node_list->push_back(first_node);
			last_index += 1;
		}
		
		try {
			second_index = index_map.at(second_node); 
		}
		catch (exception e){
			
			index_map.insert(pair<string, int>(second_node, last_index));
			
			second_index = last_index;
			node_list->push_back(second_node);
			last_index += 1;
		}
		
		try {
			neighbours = adj_map.at(second_index); 
		}
		catch (exception e){
			neighbours = new vector<int>();
			adj_map.insert(make_pair(second_index, neighbours));
		}
		neighbours->push_back(first_index);
		
		steps += 1;
	}

	myfile.close();

	int *rowstarts = new int[last_index+1];
	int  *colindices = new int[steps];
	float *values = new float[steps];
	
	int last_row_idx = 0;
	rowstarts[0] = last_row_idx;
	
	for (int i=0; i<last_index; i++){
		try {
			neighbours = adj_map.at(i);
			
			for (int j=0; j<neighbours->size(); j++) {
				colindices[last_row_idx+j] = (*neighbours)[j];
				values[last_row_idx+j] = 1;
			}
			
			last_row_idx += neighbours->size();
			rowstarts[i+1] = last_row_idx;
			
		}
		catch (exception e){
			rowstarts[i+1] = last_row_idx;
		}
	}
	
	matrix * P = (matrix*)malloc(1*sizeof(matrix));
	P->colindices = colindices;
	P->rowstarts = rowstarts;
	P->values = values;
	P->n_edges = steps;
	P->n_nodes = last_index;
	P->node_list = node_list;
	
	return P;
}

void normalize(matrix * P){
    int i,k;
    
    float *colsum = (float*)calloc(P->n_nodes, sizeof(float));
	
    for(k=0;k<P->n_nodes;k++){
	for(i=P->rowstarts[k]; i<P->rowstarts[k+1]; i++){
		colsum[P->colindices[i]] += P->values[i];
	}
    }	
    for(k=0;k<P->n_nodes;k++){
	for(i=P->rowstarts[k]; i<P->rowstarts[k+1]; i++){
		P->values[i] /= colsum[P->colindices[i]];
	}
    }

}

void printtop5rank(float* r, matrix *P, int N){
    // Find 5 largest ranks
    priority_queue<pair<float, string>, vector< pair<float, string> > ,greater< pair<float, string> > > q; 
    
    for(int i=0; i<N; i++) {
	float rank = r[i];
	string name = (*(P->node_list))[i];
		
	if(i<5){
		q.push(make_pair(rank,name));
	}
	else {
		float smallest = q.top().first;
		if(rank>smallest) {
			q.pop();
			q.push(make_pair(rank,name));
		}
	}
    }
    
    // prints top 5 rank	
    printf("5 top rank vertices: \n");
    for (int i=0; i<5; i++){
	pair<float,string> front = q.top();
	float value = front.first;
	string name = front.second;

	q.pop();
		
	cout << "Name " << name << " Rank " << value << endl;
    }
}
